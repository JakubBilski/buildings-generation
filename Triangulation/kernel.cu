#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <array>
#include <fstream>
#include <iostream>

#include "triangulation.cuh"
#include "holing.cuh"
#include "rapidjson/document.h"
#include "rapidjson/writer.h"
#include "rapidjson/stringbuffer.h"
#include <hip/hip_runtime.h>

rapidjson::Document readDocumentFromFile(std::string path)
{
	std::ifstream ifs(path);
	std::string content((std::istreambuf_iterator<char>(ifs)),
		(std::istreambuf_iterator<char>()));
	rapidjson::Document document;
	document.Parse(content.c_str());
	ifs.close();
	return document;
}
void writeDocumentToFile(rapidjson::Document& document, std::string path)
{
	rapidjson::StringBuffer buffer;
	rapidjson::Writer<rapidjson::StringBuffer> writer(buffer);
	document.Accept(writer);
	std::cout << buffer.GetString() << std::endl;
	std::ofstream ofs(path);
	ofs << buffer.GetString();
	ofs.close();
}
rapidjson::Document createExample()
{
	rapidjson::Document outputDoc;
	outputDoc.SetObject();
	rapidjson::Value plots(rapidjson::kArrayType);
	rapidjson::Value plot(rapidjson::kObjectType);
	rapidjson::Value walls(rapidjson::kArrayType);
	rapidjson::Value wall(rapidjson::kObjectType);
	rapidjson::Value meshes(rapidjson::kArrayType);
	rapidjson::Value vertexX(12.54);
	//rapidjson::Value vertexY(10.54);
	//rapidjson::Value vertexZ(8.5432);

	rapidjson::Document::AllocatorType& allocator = outputDoc.GetAllocator();

	//front and back meshes
	double frontVerticesX[8]{ 0, 100, 100, 0, 25, 50, 50, 25 };
	double frontVerticesY[8]{ 0, 0, 100, 100, 50, 50, 75, 75 };
	rapidjson::Value frontMesh(rapidjson::kObjectType);
	rapidjson::Value backMesh(rapidjson::kObjectType);
	rapidjson::Value frontVertices(rapidjson::kArrayType);
	rapidjson::Value backVertices(rapidjson::kArrayType);
	rapidjson::Value frontUvs(rapidjson::kArrayType);
	rapidjson::Value backUvs(rapidjson::kArrayType);
	for (size_t i = 0; i < 8; i++)
	{
		rapidjson::Value frontVertex(rapidjson::kObjectType);
		frontVertex.AddMember("x", rapidjson::Value().SetDouble(frontVerticesX[i]), allocator);
		frontVertex.AddMember("y", rapidjson::Value().SetDouble(frontVerticesY[i]), allocator);
		frontVertex.AddMember("z", rapidjson::Value().SetDouble(0), allocator);
		frontVertices.PushBack(frontVertex, allocator);

		rapidjson::Value frontUv(rapidjson::kObjectType);
		frontUv.AddMember("x", rapidjson::Value().SetDouble(frontVerticesX[i]/100), allocator);
		frontUv.AddMember("y", rapidjson::Value().SetDouble(frontVerticesY[i]/100), allocator);
		frontUvs.PushBack(frontUv, allocator);
		
		rapidjson::Value backVertex(rapidjson::kObjectType);
		backVertex.AddMember("x", rapidjson::Value().SetDouble(frontVerticesX[i]), allocator);
		backVertex.AddMember("y", rapidjson::Value().SetDouble(frontVerticesY[i]), allocator);
		backVertex.AddMember("z", rapidjson::Value().SetDouble(10), allocator);
		backVertices.PushBack(backVertex, allocator);

		rapidjson::Value backUv(rapidjson::kObjectType);
		backUv.AddMember("x", rapidjson::Value().SetDouble(frontVerticesX[i] / 100), allocator);
		backUv.AddMember("y", rapidjson::Value().SetDouble(frontVerticesY[i] / 100), allocator);
		backUvs.PushBack(backUv, allocator);
	}
	rapidjson::Value frontTriangles(rapidjson::kArrayType);
	rapidjson::Value backTriangles(rapidjson::kArrayType);
	rapidjson::Value frontNormals(rapidjson::kArrayType);
	rapidjson::Value backNormals(rapidjson::kArrayType);

	int triangles[24]{ 0,3,4,0,4,5,0,5,1,1,5,2,2,5,6,2,6,3,6,7,3,4,3,7 };
	for (size_t i = 0; i < 24; i++)
	{
		frontTriangles.PushBack(rapidjson::Value().SetInt(triangles[i]), allocator);
		backTriangles.PushBack(rapidjson::Value().SetInt(triangles[23-i]), allocator);
		rapidjson::Value frontNormalVector(rapidjson::kObjectType);
		rapidjson::Value backNormalVector(rapidjson::kObjectType);
		frontNormalVector.AddMember("x", rapidjson::Value().SetDouble(0), allocator);
		frontNormalVector.AddMember("y", rapidjson::Value().SetDouble(0), allocator);
		frontNormalVector.AddMember("z", rapidjson::Value().SetDouble(-1), allocator);
		backNormalVector.AddMember("x", rapidjson::Value().SetDouble(0), allocator);
		backNormalVector.AddMember("y", rapidjson::Value().SetDouble(0), allocator);
		backNormalVector.AddMember("z", rapidjson::Value().SetDouble(1), allocator);
		frontNormals.PushBack(frontNormalVector, allocator);
		backNormals.PushBack(backNormalVector, allocator);
	}

	frontMesh.AddMember("vertices", frontVertices, allocator);
	frontMesh.AddMember("triangles", frontTriangles, allocator);
	frontMesh.AddMember("normals", frontNormals, allocator);
	frontMesh.AddMember("uvs", frontUvs, allocator);
	frontMesh.AddMember("material", rapidjson::Value().SetString("plaster_blue_damaged"), allocator);

	backMesh.AddMember("vertices", backVertices, allocator);
	backMesh.AddMember("triangles", backTriangles, allocator);
	backMesh.AddMember("normals", backNormals, allocator);
	backMesh.AddMember("uvs", backUvs, allocator);
	backMesh.AddMember("material", rapidjson::Value().SetString("plaster_blue_damaged"), allocator);

	meshes.PushBack(frontMesh, allocator);
	meshes.PushBack(backMesh, allocator);
	wall.AddMember("meshes", meshes, allocator);
	walls.PushBack(wall, allocator);
	plot.AddMember("walls", walls, allocator);
	plot.AddMember("models", rapidjson::Value().SetArray(), allocator);
	plot.AddMember("type", rapidjson::Value().SetString("example"), allocator);
	plots.PushBack(plot, allocator);
	outputDoc.AddMember("plots", plots, allocator);
	return outputDoc;
}
int testTraingulation()
{

	const int duplicates = 200;
	const int noVertices = 6;
	int noVerticesInWallsBfr[duplicates + 1];
	for (size_t i = 0; i < duplicates + 1; i++)
	{
		noVerticesInWallsBfr[i] = i * noVertices;
	}
	int* d_noVerticesInWallsBfr;
	hipMalloc(&d_noVerticesInWallsBfr, sizeof(int) * (duplicates + 1));
	hipMemcpy(d_noVerticesInWallsBfr, noVerticesInWallsBfr, sizeof(int) * (duplicates + 1), hipMemcpyHostToDevice);

	//around 300 vertices for a block is too much for a shared memory
	//while 240 is fine

	const int noWallsForBlock = 40;
	const int noBlocks = (duplicates - 1) / noWallsForBlock + 1;
	int noWallsInBlocksBfr[noBlocks + 1];
	for (size_t i = 0; i < noBlocks + 1; i++)
	{
		noWallsInBlocksBfr[i] = i * noWallsForBlock;
	}
	int* d_noWallsInBlocksBfr;
	hipMalloc(&d_noWallsInBlocksBfr, sizeof(int)*(noBlocks + 1));
	hipMemcpy(d_noWallsInBlocksBfr, noWallsInBlocksBfr, sizeof(int)*(noBlocks + 1), hipMemcpyHostToDevice);

	float3 verticesInWalls[duplicates * noVertices];
	for (size_t i = 0; i < duplicates * noVertices; i += noVertices)
	{
		verticesInWalls[i].x = 0;
		verticesInWalls[i].y = 3;
		verticesInWalls[i + 1].x = 2;
		verticesInWalls[i + 1].y = 0;
		verticesInWalls[i + 2].x = 1;
		verticesInWalls[i + 2].y = 2;
		verticesInWalls[i + 3].x = 3;
		verticesInWalls[i + 3].y = 2;
		verticesInWalls[i + 4].x = 2;
		verticesInWalls[i + 4].y = 0;
		verticesInWalls[i + 5].x = 4;
		verticesInWalls[i + 5].y = 3;
	}
	float3* d_verticesInWalls;
	hipMalloc(&d_verticesInWalls, sizeof(float3) * duplicates * noVertices);
	hipMemcpy(d_verticesInWalls, verticesInWalls, sizeof(float3) * duplicates * noVertices, hipMemcpyHostToDevice);
	int sizeOfSharedMemoryPerBlock = sizeof(float3) * noWallsForBlock * noVertices + sizeof(int) * noWallsForBlock * noVertices * 7 + sizeof(int) * noWallsForBlock * 2;
	int* d_triangles;
	int trianglesSize = 3 * (noVertices - 2) * duplicates;
	hipMalloc(&d_triangles, sizeof(int) * trianglesSize);
	triangulatePolygon << <noBlocks, NO_THREADS, sizeOfSharedMemoryPerBlock >> > (d_noVerticesInWallsBfr, d_noWallsInBlocksBfr, d_verticesInWalls, noVertices*noWallsForBlock, d_triangles);
	int* triangles = (int*)malloc(sizeof(int) * trianglesSize);
	hipMemcpy(triangles, d_triangles, sizeof(int) * trianglesSize, hipMemcpyDeviceToHost);
	printf("Triangles:\n");
	for (size_t i = 0; i < trianglesSize; i += 3)
	{
		printf("%d %d %d\n", triangles[i], triangles[i + 1], triangles[i + 2]);
	}
	//rapidjson::Document& example = createExample();
	//writeDocumentToFile(example, "TriangulationOutput.json");


	return 0;
}
int testHoling()
{
	int noWalls = 1;
	int noVerticesInContoursBfr[2]{ 0, 4 };
	int noHolesInWallsBfr[2]{ 0, 1 };
	int noVerticesInHolesBfr[2]{ 0,4 };
	float3 verticesInContours[4]
	{
		{0,1,0},
		{2,0,0},
		{4,3,0},
		{2,4,0}
	};
	float3 verticesInHoles[4]
	{
		{2,1,0},
		{1,2,0},
		{2,3,0},
		{3,2,0}
	};
	float3 out_holesAndContours[10];
	mergeHolesAndContoursCPU(
		noWalls,
		noVerticesInContoursBfr,
		noHolesInWallsBfr,
		noVerticesInHolesBfr,
		verticesInContours,
		verticesInHoles,
		out_holesAndContours
	);
	for (int i = 0; i < 10; i++)
	{
		printf("%f %f\n", out_holesAndContours[i].x, out_holesAndContours[i].y);
	}
	return 0;
}


int main()
{
	testHoling();
	return 0;
}